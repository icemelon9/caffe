#include "hip/hip_runtime.h"
// ----------------------------------------------------------------------
// Written by Haichen Shen
// ----------------------------------------------------------------------

#include "caffe/layers/global_roi_layer.hpp"

namespace caffe {

namespace {

template <typename Dtype>
__global__ void global_roi_kernel(const Dtype* im_info, Dtype* out) {
  int idx = threadIdx.x;
  out[idx * 5] = (Dtype) idx;
  out[idx * 5 + 1] = 0.;
  out[idx * 5 + 2] = 0.;
  out[idx * 5 + 3] = im_info[1] - 1;
  out[idx * 5 + 4] = im_info[0] - 1;
}

template __global__ void global_roi_kernel(const float* im_info, float* out);
template __global__ void global_roi_kernel(const double* im_info, double* out);

} // namespace

template <typename Dtype>
void GlobalRoILayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top) {
  const Dtype* im_info = bottom[0]->gpu_data();
  Dtype* out = top[0]->mutable_gpu_data();
  global_roi_kernel<<<1, 1>>>(im_info, out);
}

template <typename Dtype>
void GlobalRoILayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  LOG(FATAL) << "GlobalRoILayer doesn't support back propagation";
}

INSTANTIATE_LAYER_GPU_FUNCS(GlobalRoILayer);

} // namespace caffe
